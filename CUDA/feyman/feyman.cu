
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#define NUMBER_OF_THREADS_PER_BLOCK 1024

__device__ double powCuda(double x, int) {
  return x * x;
}

int i4_ceiling(double x)
{
  int value = (int)x;
  if (value < x)
    value = value + 1;
  return value;
}

int i4_min(int i1, int i2)
{
  int value;
  if (i1 < i2)
    value = i1;
  else
    value = i2;
  return value;
}
__device__ double potential(double a, double b, double c, double x, double y, double z)
{
  return 2.0 * (powCuda(x / a / a, 2) + powCuda(y / b / b, 2) + powCuda(z / c / c, 2)) + 1.0 / a / a + 1.0 / b / b + 1.0 / c / c;
}

__device__ double r8_uniform_01(int *seed)
{
  int k;
  double r;

  k = *seed / 127773;

  *seed = 16807 * (*seed - k * 127773) - k * 2836;

  if (*seed < 0)
  {
    *seed = *seed + 2147483647;
  }
  r = (double)(*seed) * 4.656612875E-10;

  return r;
}

void timestamp(void)
{
#define TIME_SIZE 40

  static char time_buffer[TIME_SIZE];
  const struct tm *tm;
  time_t now;

  now = time(NULL);
  tm = localtime(&now);

  strftime(time_buffer, TIME_SIZE, "%d %B %Y %I:%M:%S %p", tm);

  printf("%s\n", time_buffer);

  return;
#undef TIME_SIZE
}



__global__ void feyman_device(int N, int ni, int nj, int nk, double a, double b, double c, double* err, int* n_inside, double* wt) {

  int index = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;

  int i = blockIdx.x + 1;
  int j = blockIdx.y + 1;
  int k = blockIdx.z + 1;

  double x = ((double)(ni - i) * (-a) + (double)(i - 1) * a) / (double)(ni - 1);
  double y = ((double)(nj - j) * (-b) + (double)(j - 1) * b) / (double)(nj - 1);
  double z = ((double)(nk - k) * (-c) + (double)(k - 1) * c) / (double)(nk - 1);

  double chk;
  double dx;
  double dy;
  double dz;
  const double h = 0.001;
  double stepsz;
  int seed = 123456789 + threadIdx.x;
  int steps;
  int trial;
  double us;
  double ut;
  double vh;
  double vs;
  double x1;
  double x2;
  double x3;
  double w;
  double w_exact;
  double we;
  steps = 0;

  const int dim = 3;
  stepsz = sqrt((double)dim * h);

  chk = powCuda(x / a, 2) + powCuda(y / b, 2) + powCuda(z / c, 2);
  
  if (1.0 < chk)
  {
    w_exact = 1.0;
    wt[index] = 1.0;
    return;
  }

  // umesto svi na jednu, delovi na jednu
  if (threadIdx.x == 0) {
    atomicAdd(n_inside, 1);
  }
  
  w_exact = exp(powCuda(x / a, 2) + powCuda(y / b, 2) + powCuda(z / c, 2) - 1.0);

  double wt_tmp = 0;
  for (trial = threadIdx.x; trial < N; trial += NUMBER_OF_THREADS_PER_BLOCK)
  {
    x1 = x;
    x2 = y;
    x3 = z;
    w = 1.0;
    chk = 0.0;
    while (chk < 1.0)
    {
      ut = r8_uniform_01(&seed);
      if (ut < 1.0 / 3.0)
      {
        us = r8_uniform_01(&seed) - 0.5;
        if (us < 0.0)
          dx = -stepsz;
        else
          dx = stepsz;
      }
      else
        dx = 0.0;

      ut = r8_uniform_01(&seed);
      if (ut < 1.0 / 3.0)
      {
        us = r8_uniform_01(&seed) - 0.5;
        if (us < 0.0)
          dy = -stepsz;
        else
          dy = stepsz;
      }
      else
        dy = 0.0;

      ut = r8_uniform_01(&seed);
      if (ut < 1.0 / 3.0)
      {
        us = r8_uniform_01(&seed) - 0.5;
        if (us < 0.0)
          dz = -stepsz;
        else
          dz = stepsz;
      }
      else
        dz = 0.0;

      vs = potential(a, b, c, x1, x2, x3);
      x1 = x1 + dx;
      x2 = x2 + dy;
      x3 = x3 + dz;

      steps++;

      vh = potential(a, b, c, x1, x2, x3);

      we = (1.0 - h * vs) * w;
      w = w - 0.5 * h * (vh * we + vs * w);

      chk = powCuda(x1 / a, 2) + powCuda(x2 / b, 2) + powCuda(x3 / c, 2);
    }  
    wt_tmp += w;
  }

  atomicAdd(&(wt[index]), wt_tmp); 

  //  sacekaj ih pa samo jedna da izdeli
  __syncthreads();
  if (threadIdx.x == 0) {
    wt[index] = wt[index] / (double)(N);
  
    atomicAdd(err, powCuda(w_exact - wt[index], 2));
  }
}

// print na stdout upotrebiti u validaciji paralelnog resenja
int main(int arc, char **argv)
{
  double a = 3.0;
  double b = 2.0;
  double c = 1.0;
  int dim = 3;
  double err;
  double h = 0.001;
  int n_inside;
  int ni;
  int nj;
  int nk;
  double stepsz;

  int N = atoi(argv[1]);
  timestamp();

  printf("A = %f\n", a);
  printf("B = %f\n", b);
  printf("C = %f\n", c);
  printf("N = %d\n", N);
  printf("H = %6.4f\n", h);

  stepsz = sqrt((double)dim * h);

  if (a == i4_min(i4_min(a, b), c))
  {
    ni = 6;
    nj = 1 + i4_ceiling(b / a) * (ni - 1);
    nk = 1 + i4_ceiling(c / a) * (ni - 1);
  }
  else if (b == i4_min(i4_min(a, b), c))
  {
    nj = 6;
    ni = 1 + i4_ceiling(a / b) * (nj - 1);
    nk = 1 + i4_ceiling(c / b) * (nj - 1);
  }
  else
  {
    nk = 6;
    ni = 1 + i4_ceiling(a / c) * (nk - 1);
    nj = 1 + i4_ceiling(b / c) * (nk - 1);
  }

  err = 0.0;
  n_inside = 0;

  dim3 dimGrid(ni, nj, nk);
  dim3 dimBlock(NUMBER_OF_THREADS_PER_BLOCK);

  int* n_inside_device;
  double* err_device;
  double* wt_device;


  hipMalloc((void**)&n_inside_device, sizeof(int));
  hipMalloc((void**)&err_device, sizeof(double));
  hipMalloc((void**)&wt_device, ni * nj * nk * sizeof(double));

  feyman_device<<< dimGrid, dimBlock >>>(N, ni, nj, nk, a, b, c, err_device, n_inside_device, wt_device); 

  // kopiranje mi sinhronizuje
  hipMemcpy(&n_inside, n_inside_device, sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(&err, err_device, sizeof(double), hipMemcpyDeviceToHost);

  hipFree(n_inside_device);
  hipFree(err_device);
  hipFree(wt_device);
  
  err = sqrt(err / (double)(n_inside));

  printf("\n\nRMS absolute error in solution = %e\n", err);
  timestamp();

  return 0;
}
